#include "hip/hip_runtime.h"
/* MAD Filter on GPU

Version 1.0
Runs on single bin size
Input:  filename
        Number of samples to filter
        Bin size
        Threshold (multiple of sigma)
        Option for filtering
	Header size in bytes
 
Basic version using histogram method for median. 24/01/13
Replace with random numbers. Added 28/01/13
Finding mean and rms before and after filtering. Added 02/02/13 
Copy back only flags file (bool)

Compile it with following line:
nvcc -Xptxas="-v" -o madfilter_small madfilter_small.cu -arch=sm_20 

(Rohini Joshi, 2013 - rmjoshi.06@gmail.com)

*/

/* Modified the code to work with 8-bit unsigned data 

 Kaustubh Rajwade (Sept 2017 - Manchester)*/
  
#include<hip/hip_runtime.h>
#include<hiprand.h>  		// random num generation in cuda
#include<hiprand/hiprand_kernel.h>       // random num generation in cuda
#include<stdio.h>
#include<sys/time.h>
#include<iostream>
#include<math.h>
#include<string.h>
#include "madfilter_small.h"


// __device__ --> is a dev fn to be run on GRID and can be called only from kernel or device fn

__device__ float randomnumber(int t, int i){

hiprandState s; 
float x;
// hiprand_init() sets up an initial state s. with seed t(thread id) and sequence number 0 and offset i
// Each bin is filtered with a separate thread. Thus normal distribution of random numbers is preserved within a bin
hiprand_init(t, 0, i, &s);  // t is a seed, i is offset in seq of random numbers 
// Generate random number from normal distribution
x = hiprand_normal(&s);

return x;
}

__global__ void madfilter( int *d_data, int binsize, int bins, int op, float *dev, int *not_flagged_data, bool *d_flag, float *d_rms_b, float *d_rms_a, float *d_mad, int mult, time_t currTime){
    
// {0} initialised the whole array. blockDim = number of threads/block=32, tid indexes all threads in the grid
// everything below runs for each thread through threadIdx.x and blockIdx.x
int i, j=0, c=0,d,flag=0,odd=0,sum=0, sumsq=0,histdev[256] = {0},hist[256] = {0}, tid = threadIdx.x + blockIdx.x * blockDim.x;
int lw = tid * (binsize);   // the index in the original data array for each bin beginning
int up = lw + (binsize);    //                    same                           end
float mean, med, mad, thresh;

// variable j is to store effective size of bin (after flagging extremities)
if (tid < bins){    // end crap is not accessed

/* Flagging and generating histogram */
for ( i=lw; i<up; i++){
	sum += d_data[i];
	sumsq += d_data[i]*d_data[i];

	// Flag extremities
        if((d_data[i]==0) || (d_data[i] == 255)){
                continue;
        }else{
                hist[d_data[i]] += 1;
                not_flagged_data[lw+j] = d_data[i];
		j+=1;
        }
}
/* Find RMS before filtering */
mean = sum/(binsize);
d_rms_b[tid] = sqrtf( sumsq/(binsize) - mean*mean );
sum = 0;sumsq = 0;
/* Find median. Two methods for even/odd sizes. Modify if data is 4 bit
flag = 1/0 if median is floating point/int
odd = 1/0 if data set is odd/even 
median can only be float if data set is even */

if (j%2 == 0){
        d = j/2;
        for ( i=0; i<(256); i++){
                c = c + hist[i];
                if (c==d){
                        med =(float)( (2*(i) + 1)*0.5 );
                        flag = 1;
                        break;
                }else if (c>d){
                        med = i;
                        break;
                }else
                        continue;
        }
}else{
        d = (j + 1)/2;
        odd = 1;
        c = 0;
        for ( i=0; i<(256); i++){
                c = c + hist[i];
                if (c >= d){
                        med = i;
                        break;
                }
        }
}
//  MAD
int s = 0, ii;
if (flag == 0){
        for ( i=lw; i<lw+j; i++){
                dev[i] = fabs( not_flagged_data[i] - med );
                ii = (int)(ceil(dev[i]));
		histdev[ii] += 1;
        }
        /* two submethods for even/odd data sets */
        if (odd == 0){
        for ( i=0; i<(256); i++){
                s = s+histdev[i];
                if (s == d){
                        mad = (float)( (2*(i) + 1)*0.5 );
                        break;
                }else if (s > d ){
                        mad = i;
                        break;
                }else
                        continue;
        }
        }else{
        for ( i=0; i<(256); i++){
                s = s + histdev[i];
                if(s >= d){
                        mad = i;
                        break;
                }
        }
        }
}else{
        int p;
        for ( i=lw; i<lw+j; i++){
                dev[i] = (float)fabs( not_flagged_data[i] - med );
                p = (int) dev[i];
                histdev[p] += 1;
        }
        int s = 0;
        d = j/2;
        for ( i=0; i<(256); i++){
                s = s+histdev[i];
                if (s == d){
                        mad = (float)( (2*(i) + 1)*0.5 + 0.5 );
                        break;
                }else if (s > d){
                        mad = (float)( i + 0.5 );
                        break;
                }else
                        continue;
        }

}
  
d_mad[tid] = mad;
thresh = mult*1.4826*mad;
//filtering

// thresh = mult*1.48*mad
// if abs(d-med) > thresh ---> flag

for( i=lw; i<up; i++){
	if ( (fabsf(d_data[i]-med) > thresh) || (d_data[i] == 0) || (d_data[i] == 255)  ){
        	if(op == 0){
	                d_data[i] = 0;
	        }else if(op == 1){
        	        d_data[i] = med;
		}else if(op == 2){
			d_data[i] = rint(mean + 1.4826*mad*randomnumber(currTime, i-lw));
	        }else if(op == 1){
			d_data[i] = thresh;
		}d_flag[i] = 0;
	}
	else{
		d_flag[i] = 1;
	}
	sum += d_data[i];
	sumsq += d_data[i]*d_data[i];
}

/* Find RMS after filtering */
mean = sum/(binsize);
d_rms_a[tid] = sqrtf( sumsq/(binsize) - mean*mean );

/*if(*op == 0){
	printf("replace with zero\n");
}else if(*op == 1){ 
	printf("replace with median\n");
}else if(*op == 2){    
         printf("replace with random number\n");
}
*/

}
}

 
void run_madfilter(unsigned char* h_cdata, int size, int bsize){

int i,ind=0, mult_thresh, bins;
int *h_data, *d_data, *not_flagged_data, op_int = 1;
float *h_rms_b, *h_rms_a, *d_rms_b, *d_rms_a, *h_mad, *d_mad, *dev;
float time_initial_dev, time_kernel, time_copyback;
bool *h_flag, *d_flag;

//Define and create CUDA events start and stop for timing GPU activity
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipSetDevice(0);
// Number of whole bins that can be filtered in the dataset
bins = (int)size/bsize;
// size is now made a multiple of the bin size
size = bins*bsize;


/* Allocate and store input on host */
h_data = (int *)malloc(size*sizeof(int));	// actual data - will be read from SHM
h_rms_b = (float *)malloc(bins*sizeof(float));	// RMS before filtering for each bin    - for checking
h_rms_a = (float *)malloc(bins*sizeof(float));	// RMS after filtering for each bin    - for checking
h_mad = (float *)malloc(bins*sizeof(float));	// MAD value for each bin    - for checking
h_flag = (bool *)malloc(size*sizeof(bool));	// Flags 



// Reading data as 4 byte integers
for(i = 0; i < size; i++) {
          h_data[ind] = (int) h_cdata[i];
	  ++ind;
        }

/* Allocate i/o and store input on device */
hipEventRecord( start, 0 ); // Start CUDA timer
hipMalloc( (void **)&d_data, size*sizeof(int) );
hipMalloc( (void **)&d_rms_b, bins*sizeof(float) );    // dont need this
hipMalloc( (void **)&d_rms_a, bins*sizeof(float) );    // dont need this
hipMalloc( (void **)&d_mad, bins*sizeof(float) );
hipMalloc( (void **)&dev, bins*sizeof(float)*bsize );
hipMalloc( (void **)&not_flagged_data, bins*sizeof(int)*bsize );
hipMalloc( (void **)&d_flag, size*sizeof(bool) );	// actual flags, can be bool
hipMemcpy( d_data, h_data, size*sizeof(int), hipMemcpyHostToDevice );

hipEventRecord( stop, 0 );	// Stop and store time elapsed
hipEventSynchronize(stop);
hipEventElapsedTime( &time_initial_dev, start, stop);

/* Setup grid and run kernel */
int blocks, threads = 32;
blocks = (bins + threads - 1)/threads;	// Mathematically equivalent to a ceil(bins/threads) = number of blocks so that 1 thread/bin
//printf("Grid dim [%d 1] Block dim [%d 1]\n", blocks, threads);

hipEventRecord(start, 0);   // to start timing

// Seeding using current time

time_t currTime = time(NULL);


mult_thresh = 3.0; // MAD filter threshold
// send # of blocks and threads to the cuda kernel. dev is y-median(y), d_flag is array of bools, mult_thresh=3
// is asynchronous => comes back to cpu even before finishing
madfilter<<<blocks, threads>>>( d_data, bsize, bins, op_int, dev, not_flagged_data, d_flag, d_rms_b, d_rms_a, d_mad, mult_thresh, currTime);

hipDeviceSynchronize();
hipEventRecord(stop, 0);
hipEventSynchronize(stop);   // makes sure gpu is done, and is part of the timing module. but can synchronise in other ways instead 
			      //if not insterested in timing
hipEventElapsedTime( &time_kernel, start, stop);


//printf("Number of Bins = %d\n", bins);
//printf("Time for executing kernel = %f msec\n", time_kernel);

/* Copy data back to host */
hipEventRecord(start, 0);

hipMemcpy( h_flag, d_flag, size*sizeof(bool), hipMemcpyDeviceToHost );
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime( &time_copyback, start, stop);
hipMemcpy( h_mad, d_mad, bins*sizeof(float), hipMemcpyDeviceToHost );
hipMemcpy( h_data, d_data, size*sizeof(int), hipMemcpyDeviceToHost );
hipMemcpy( h_rms_b, d_rms_b, bins*sizeof(float), hipMemcpyDeviceToHost );
hipMemcpy( h_rms_a, d_rms_a, bins*sizeof(float), hipMemcpyDeviceToHost );

// Free memory on the device
hipFree(d_data);
hipFree(d_mad);
hipFree(d_rms_b);
hipFree(d_rms_a);
hipFree(dev);
hipFree(not_flagged_data);
hipFree(d_flag);
hipEventDestroy(start); 
hipEventDestroy(stop); 
//printf("Time to copyback = %f ms", time_copyback);
//printf("Total time = %f ms", time_copyback + time_kernel) ;

ind =0;
for (i=0;i<size;i++){
  h_cdata[i] = (h_data[ind] & (255));
  ++ind;
}

// Free all arrays on the host
free(h_data);
free(h_rms_a);
free(h_rms_b);
free(h_flag);
free(h_mad);

//printf("Data copied back to host\n");
return;

}
