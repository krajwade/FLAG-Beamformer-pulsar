#include "hip/hip_runtime.h"
#include "cuda_utils.h"

/* Kernel for quantization */
__global__ void Quant(float pfbuf[], int pibuf[] , int n, float fMax, float fMin) {
   /* blockDim.x = threads_per_block                            */
   /* First block gets first threads_per_block components.      */
   /* Second block gets next threads_per_block components, etc. */
   int i = blockDim.x * blockIdx.x + threadIdx.x;

   float frange;
   float fIntMax = (float) (powf(2.0 , 8) - 1.0);
   frange = fMax - fMin;
   /* block_count*threads_per_block may be >= n */
   if (i < n) {
  	 pibuf[i] = (int) roundf(((pfbuf[i]-fMin)/frange)*fIntMax);
   }
}  /* Quant */


__global__ void reverseArrayBlockFloat(float *d_b , float *d_a )
{
 int bx = blockIdx.x , tx = threadIdx.x ; 
 int old_id = blockDim.x * bx+ tx ;


// GridDim.x gives no. of block in grid in X dimension
 int new_id = (blockDim.x * gridDim.x) - 1 -  old_id ; 

 
 d_b[old_id] = d_a[new_id ]; 

}

__global__ void reverseArrayBlockNewFloat(float *d_b , float *d_a, int binsize, int bins )
{
 int bx = blockIdx.x , tx = threadIdx.x ; 
 int tid = blockDim.x * bx+ tx ;
 if (tid < bins)
 {
 int lw = tid * (binsize);
 int uw = lw + binsize;
 int i;
 for (i = 0; i < binsize; i++)
 {
   d_b[i + lw] = d_a[uw -1 - i];
 } 

 }
}


/* Qunatization GPU-based. Still debugging! */
void run_quant(float* h_pfbuf,int nchans, unsigned char* h_pcbuf) 
{
   int i;
   float fMin,fMax;
   int *d_pibuf, *h_pibuf;
   float *d_pfbuf;
   int threads_per_block;
   int block_count;
   size_t size_f,size_i;

   size_f = nchans*sizeof(float);
   size_i = nchans*sizeof(int);
   
  /* find Min and max */

   for (i = 0; i < nchans; ++i){
      if (h_pfbuf[i] > fMax)
           fMax = h_pfbuf[i];
      if (h_pfbuf[i] < fMin)
           fMin = h_pfbuf[i];

   }
   i = 0;
   /* Allocate vectors in device memory */
   h_pibuf = (int*)malloc(nchans * sizeof(int));
   hipMalloc(&d_pfbuf, size_f);
   hipMalloc(&d_pibuf, size_i);

   float* ptr = h_pfbuf;

   /* Copy vectors from host memory to device memory */
   hipMemcpy(d_pfbuf, ptr, size_f, hipMemcpyHostToDevice);

   checkCUDAError("memcpy");
   /* Define block size */
   threads_per_block = 250;

   /* Define grid size.  If we just computed n/threads_per_block */
   /* we might get fewer threads than vector components.  Using  */
   /* ceil(n/threads_per_block) guarantees at least one thread   */
   /* per vector component.  The following formula is a kludge   */
   /* since it appears that the CUDA ceil function doesn't work  */
   /* correctly.                                                 */
   block_count = (nchans + threads_per_block - 1)/threads_per_block;

   /* Invoke kernel using block_count blocks, each of which  */
   /* contains threads_per_block threads                     */
   Quant<<<block_count, threads_per_block>>>(d_pfbuf, d_pibuf, nchans, fMax,fMin);

   /* Wait for the kernel to complete */
   hipDeviceSynchronize();

  
   checkCUDAError("kernel Invocation");

   unsigned char* cptr = h_pcbuf;
   /* Copy result from device memory to host memory */
   /* h_z contains the result in host memory        */
   hipMemcpy(h_pibuf, d_pibuf, size_i, hipMemcpyDeviceToHost);

   checkCUDAError("memcpy");

   for (i = 0; i < nchans; ++i)
   {
	*(cptr) = (h_pibuf[i] & (255));
	++cptr;
   }
   
   /* Free device memory */
   hipFree(d_pfbuf);
   hipFree(d_pibuf);
   /* Free host memory */
   free(h_pibuf); 
   return;
} 

void channel_flip_float( float *h_a, int size, int nchans)
{
    
    // pointer for device memory
    float *d_b, *d_a;
    float time_kernel;
    // define grid and block size
    int numThreadsPerBlock = 10;

    int bins = (int)size/nchans;
    // Part 1 of 2: compute number of blocks needed based on array size and desired block size
    int numBlocks = (bins + numThreadsPerBlock -1)/numThreadsPerBlock ;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // allocate host and device memory
    size_t memSize = size * sizeof(float);
    hipMalloc( (void **) &d_a, memSize );
    hipMalloc( (void **) &d_b, memSize );

    // Initialize input array on host

    hipEventRecord(start, 0);   // to start timing
    // Copy host array to device array
    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice );



    // launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
 
    reverseArrayBlockNewFloat<<< dimGrid, dimBlock >>>( d_b, d_a, nchans, bins );

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");

    // device to host copy
    hipMemcpy( h_a, d_b, memSize, hipMemcpyDeviceToHost );

    // Check for any CUDA errors
    checkCUDAError("memcpy");

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);   // makes sure gpu is done, and is part of the timing module. but can synchronise in other ways instead 
    //if not insterested in timing
    hipEventElapsedTime( &time_kernel, start, stop);
    //printf("Time for executing channel flip = %f msec\n", time_kernel);


    // free device memory
    hipFree(d_a);
    hipFree(d_b);

    return;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

